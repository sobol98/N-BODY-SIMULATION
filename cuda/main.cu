#include "hip/hip_runtime.h"
/*
    * File path: 
    * .../N-body-simulation/cuda
    * 
    * To compile:
    * nvcc main.cu -o main
    * 
    * To run:
    * ./main N 
    * where N is number of bodies
*/


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


//---------- Constants ----------//

#define G 6.67430e-11
#define DELTA_TIME 0.01 // time step in simulation time (in seconds)
#define T_END 100000 // how many seconds (in real time) the simulation will run
// #define N 10 // number of bodies

#define BLOCK_SIZE 256 // idk what this is


// CUDA error checking
#define CUDACHECK(err) do { cuda_check((err), __FILE__, __LINE__); } while(false)
inline void cuda_check(hipError_t error_code, const char *file, int line)
{
    if (error_code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error %d: %s. In file '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
        fflush(stderr);
        exit(error_code);
    }
}


//---------- Structs ----------//
struct Body {
    double3 position;
    double3 velocity;
    double3 force;
    float mass;
};



//---------- Functions ----------//

__device__ double dot_product(double3 a) {
    return a.x * a.x + a.y * a.y + a.z * a.z;
}

//replace nan value with 0
__device__ void check_and_replace_nan(double* value) {
    if (isnan(*value)) {
        *value = 0.0f; 
    }
}


// this function calculate initial position of the N bodies in the our empty universum
void initBodies(Body *bodies, int n) {
    float destination_parameter = 1.0e+3;
    float velocity_parameter = 1.0e+0;
    float mass_parameter = 1.0e+24;


    for (int i = 0; i < n; i++) {
        bodies[i].position.x = ((rand() % 1000) - 500)*destination_parameter;
        bodies[i].position.y = ((rand() % 1000) - 500)*destination_parameter;
        bodies[i].position.z = ((rand() % 1000) - 500)*destination_parameter;
        
        bodies[i].velocity.x = ((rand() % 1000) - 500)*velocity_parameter;
        bodies[i].velocity.y = ((rand() % 1000) - 500)*velocity_parameter;
        bodies[i].velocity.z = ((rand() % 1000) - 500)*velocity_parameter;
        
        bodies[i].mass = (rand() % 1000 + 1) * mass_parameter;                                       
    }
}

//CUDA kernel that calculates the gravitational forces acting on each body
__global__ void calculate_parameters(Body *bodies, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // int i =0;
    if (i < n) {
        double3 f;
        f.x =0.0f;
        f.y =0.0f;
        f.z =0.0f;

        // f = make_double3(0.0, 0.0, 0.0);

        for (int j = 0; j < n; j++) {
            if (i != j) {
                double3 diff;
                diff.x = bodies[j].position.x - bodies[i].position.x;
                diff.y = bodies[j].position.y - bodies[i].position.y;
                diff.z = bodies[j].position.z - bodies[i].position.z;

                double dist = sqrtf(dot_product(diff)); //calculation of the length of the displacement vector (diagonal of 3 dimensions)
                double forceMagnitude = G * bodies[i].mass * bodies[j].mass / (dist * dist + 1e-10f);  //+ 1e-10f -> prevent division by zero
                
                f.x += forceMagnitude * diff.x / dist;
                f.y += forceMagnitude * diff.y / dist;
                f.z += forceMagnitude * diff.z / dist;
            }
        }
        bodies[i].force = f;
    }
}

//CUDA kernel that updates the positions and velocities of each body based on the forces calculated.
__global__ void updateBodies(Body *bodies, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        bodies[i].velocity.x += bodies[i].force.x / bodies[i].mass * DELTA_TIME;
        bodies[i].velocity.y += bodies[i].force.y / bodies[i].mass * DELTA_TIME;
        bodies[i].velocity.z += bodies[i].force.z / bodies[i].mass * DELTA_TIME;

        bodies[i].position.x += bodies[i].velocity.x * DELTA_TIME/2.0f;
        bodies[i].position.y += bodies[i].velocity.y * DELTA_TIME/2.0f;
        bodies[i].position.z += bodies[i].velocity.z * DELTA_TIME/2.0f;


        check_and_replace_nan(&bodies[i].force.x);
        check_and_replace_nan(&bodies[i].force.y);
        check_and_replace_nan(&bodies[i].force.z);

        check_and_replace_nan(&bodies[i].position.x);
        check_and_replace_nan(&bodies[i].position.y);
        check_and_replace_nan(&bodies[i].position.z);
    }
}

void save_results(Body *bodies, int n){ //}, char filename){

    FILE *file;
    file = fopen("results.txt", "a");


    //check if the file was opened
    if(file == NULL){ 
        printf("Error opening file\n");
        exit(1);
    }

    // format of the result: body_number mass position_x position_y position_z
    for(int i = 0; i < n; i++){
        fprintf(file, "%d %f %f %f %f\n",i, bodies[i].mass, bodies[i].position.x, bodies[i].position.y, bodies[i].position.z);
    }
    fclose(file);

    // printf("Results saved to results.txt\n");
}




int main(int argc, char **argv) {
    //remove the file if it already exists
    remove("results.txt");


    srand(time(NULL));

    if (argc != 2){
        printf("Error: wrong number of arguments\n");
        exit(1);
    }

    if (argc < 2) {
        fprintf(stderr, "Usage: %s <number of bodies>\n", argv[0]);
        exit(1);
    }

    int n = atoi(argv[1]); // Convert command-line argument to integer
    
    if (n <= 0) {
        fprintf(stderr, "Number of bodies must be a positive integer\n");
        exit(1);
    }


    // Get the number of CUDA devices
    int deviceCount;

    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA devices available\n");
        return 1;
    }

    // Query CUDA device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);                                      // Assuming using the first device
    // printf("Device Name: %s\n", prop.name);                                 //is an ASCII string identifying the device
    // printf("Number of multiprocessors: %d\n", prop.multiProcessorCount);    // is the number of multiprocessors on the device
    // printf("Total global memory: %zu bytes\n", prop.totalGlobalMem);        //is the total amount of global memory available on the device in bytes
    // printf("Maximum number of threads per block: %d bytes\n", prop.maxThreadsPerBlock);   // is the maximum number of threads per block;
    // printf("maximum size of each dimension of a block: %d bytes\n", prop.maxThreadsDim[3]);      // contains the maximum size of each dimension of a block;
    // printf("maximum size of each dimension of a grid: %d bytes\n", prop.maxGridSize[3]);        // contains the maximum size of each dimension of a grid;
    // printf("clock frequency in kilohertz: %d bytes\n", prop.clockRate);             //is the clock frequency in kilohertz;
    

    // Define CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start recording
    hipEventRecord(start);

    Body *h_bodies = (Body*)malloc(n * sizeof(Body));  // Host bodies - CPU
    Body *d_bodies;  // Device bodies - GPU

    initBodies(h_bodies, n);

    // hipMemcpy(d_bodies, h_bodies, n * sizeof(Body), hipMemcpyHostToDevice);


    CUDACHECK(hipMalloc(&d_bodies, n * sizeof(Body)));
    CUDACHECK(hipMemcpy(d_bodies, h_bodies, n * sizeof(Body), hipMemcpyHostToDevice));


    

    // checkCudaError(hipMalloc(&d_bodies, n * sizeof(Body)));
    // checkCudaError(hipMalloc(&d_forces, n * sizeof(float3)));
    // checkCudaError(hipMemcpy(d_bodies, h_bodies, n * sizeof(Body), hipMemcpyHostToDevice));

    // dim3 blockSize(BLOCK_SIZE);
    // dim3 gridSize((n + blockSize.x - 1) / blockSize.x);


    int blockSize=BLOCK_SIZE;
    int blocks = (n + blockSize - 1) / blockSize;


    double start_time = clock();

    for (int iter = 0; iter < (T_END); iter++) {
        calculate_parameters<<<blocks, blockSize>>>(d_bodies, n);
        CUDACHECK(hipDeviceSynchronize());

        updateBodies<<<blocks, blockSize>>>(d_bodies, n);
        CUDACHECK(hipDeviceSynchronize());



        CUDACHECK(hipMemcpy(h_bodies, d_bodies, n * sizeof(Body), hipMemcpyDeviceToHost));
        save_results(h_bodies, n);


        // calculate_parameters<<<gridSize, blockSize>>>(d_bodies, n); // Pass n as an argument
        // updateBodies<<<gridSize, blockSize>>>(d_bodies, n); // Pass n as an argument
        // checkCudaError(hipDeviceSynchronize());
    }


    // hipMemcpy(h_bodies, d_bodies, n * sizeof(Body), hipMemcpyHostToDevice);

    double end_time = clock();
    // Stop recording
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate and print the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken for execution: %f milliseconds\n", milliseconds);

    printf("Time taken for execution: %f seconds\n", (end_time - start_time) / CLOCKS_PER_SEC);
    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy data back to host for output
    // checkCudaError(hipMemcpy(h_bodies, d_bodies, n * sizeof(Body), hipMemcpyDeviceToHost));
    
    hipFree(d_bodies);


    free(h_bodies);

    return 0;
}